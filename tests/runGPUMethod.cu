#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include "../src/loadImage.hpp"
#include "../src/cpuFuncs.hpp"
#include "../src/gpuFuncs.hpp"
#include "../src/constants.hpp"

int main(){

  // Declaration of necessary variables
  int Nx[3] = {512, 1024, 2048};
  int Ny[3] = {512, 1024, 2048};
  clock_t begin, end;
  double elapsed;
  int manyTimes = 1;
  std::string imgPath;

  // We make a speed test of the convolutons for each image size
  for (int s = 2; s < 3; s++){

    switch (s) {
      case 0:
        imgPath = "../data/512/img1.txt";
        break;
      case 1:
        imgPath = "../data/1024/img1.txt";
        break;
      case 2:
        imgPath = "../data/2048/img1.txt";
    }

    std::cout << "***********************************************" << std::endl;
    std::cout << Nx[s] << "x" << Ny[s] << std::endl;
    std::cout << "***********************************************" << std::endl;

    std::vector<short> image= loadImage(imgPath,Nx[s],Ny[s]);
    std::vector<short> result(Nx[s]*Ny[s]);


    begin = clock();
    for (int i = 0; i < manyTimes; i++){
      GPU_convolution_tiling(image, result, Nx[s], Ny[s], gaussian);
    }
    end = clock();
    elapsed = double(end - begin) / CLOCKS_PER_SEC;
    std::cout << "GPU 1x"<< tilingFactor << " tiling:  5x5 convolution on " << manyTimes << " " << Nx[s] << "x" << Ny[s]
      <<  " images: " << elapsed << std::endl;

/*
    begin = clock();
    for (int i = 0; i < manyTimes; i++){
      GPU_convolution_tiling(image, result, Nx[s], Ny[s], laplacian);
    }
    end = clock();
    elapsed = double(end - begin) / CLOCKS_PER_SEC;
    std::cout << "GPU 1x"<<tilingFactor<<" Tiling:  3x3 convolution on " << manyTimes << " " << Nx[s] << "x" << Ny[s]
      <<  " images: " << elapsed << std::endl;
*/
  }

  return 0;
}
